#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "graph.h"
#include "MyStack.h"
#include <stdio.h>
#include <unistd.h>
using namespace std;
using namespace std::chrono;
#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define White 0
#define Gray 1
#define Black 2
#define INF (1 << 31) - 1
#define bitMapSize 10000000
#define StackSize 1000

#define INF 0x3f3f3f3f

void printlist(int *a, int size)
{
    // int size = sizeof(&a)/sizeof(a[0]); only use for array() not for int*
    // std::cout << "sizeof a :" << size << std::endl;
    for (int i = 2; i < size; i++)
    {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;
}

void printAdj(int **Adjacency, int numofVertex)
{
    // int size = sizeof(&a)/sizeof(a[0]); only use for array() not for int*
    std::cout << "numofVertex a :" << numofVertex << std::endl;

    for (int i = 0; i < numofVertex; i++)
    {
        cout<<i<<" -> ";
        int size = Adjacency[i][1];
        printlist(Adjacency[i], size + 2);
    }
    std::cout << std::endl;
}
void CreateAdjacency(int **Adjacency, int numofVertex, graph<long, long, /*int*/ long, long, long, /* char*/ long> *ginst)
{
    Adjacency[3] = new int[5];
    std::cout << ginst->vert_count << " size CreateAdjacency...\n";
    for (int i = 0; i < ginst->vert_count; i++)
    {
        int beg = ginst->beg_pos[i];
        int end = ginst->beg_pos[i + 1];
        int numofneighbor = end - beg;
        int size = numofneighbor + 2;
        // std::cout << i << " size= " << size<< "'s neighor list: \n";
        Adjacency[i] = new int[size];
        // std::cout<<" i "<<i<<"  ";
        Adjacency[i][0] = 0;
        Adjacency[i][1] = numofneighbor; //default color is 0

        if (numofneighbor > 0)
        {
            for (int j = beg; j < end; j++)
            {
                Adjacency[i][j - beg + 2] = ginst->csr[j];
                // std::cout << "j= " << j << " " << ginst->csr[j] << " ";
            }
        }
        // std::cout << "\n";
        // printlist(Adjacency[i], size);
    }
    std::cout << " \nCreateAdjacency finish \n";
}

void initAdj(int **&Adjacency, int numofVertex)
{
    std::cout << "initAdj... numofVertex :" << numofVertex << std::endl;

    for (int i = 0; i < numofVertex; i++)
    {
        Adjacency[i][0] = 0;
        // printlist(Adjacency[i], size+2);
    }
    std::cout << std::endl;
}
//-----------------------------------------------------------------------------
// MurmurHash3 was written by Austin Appleby, and is placed in the public
// domain. The author hereby disclaims copyright to this source code.
// Note - The x86 and x64 versions do _not_ produce the same results, as the
// algorithms are optimized for their respective platforms. You can still
// compile and run any of them on any platform, but your performance with the
// non-native version will be less than optimal.

__forceinline__  __host__ __device__ uint32_t rotl32( uint32_t x, int8_t r ) {
    return (x << r) | (x >> (32 - r));
  }
  
  __forceinline__ __host__ __device__ uint32_t fmix32( uint32_t h ) {
    h ^= h >> 16;
    h *= 0x85ebca6b;
    h ^= h >> 13;
    h *= 0xc2b2ae35;
    h ^= h >> 16;
    return h;
  }
  
  __forceinline__  __host__ __device__ uint32_t hash_murmur(const int64_t& key) {
  
    constexpr int len = sizeof(int64_t);
    const uint8_t * const data = (const uint8_t*)&key;
    constexpr int nblocks = len / 4;
    uint32_t h1 = 0;
    constexpr uint32_t c1 = 0xcc9e2d51;
    constexpr uint32_t c2 = 0x1b873593;
    //----------
  
    // body
    const uint32_t * const blocks = (const uint32_t *)(data + nblocks*4);
    for(int i = -nblocks; i; i++)
    {
      uint32_t k1 = blocks[i];
      k1 *= c1;
      k1 = rotl32(k1,15);
      k1 *= c2;
      h1 ^= k1;
      h1 = rotl32(h1,13); 
      h1 = h1*5+0xe6546b64;
    }
    //----------
    // tail
    const uint8_t * tail = (const uint8_t*)(data + nblocks*4);
    uint32_t k1 = 0;
    switch(len & 3)
    {
      case 3: k1 ^= tail[2] << 16;
      case 2: k1 ^= tail[1] << 8;
      case 1: k1 ^= tail[0];
              k1 *= c1; k1 = rotl32(k1,15); k1 *= c2; h1 ^= k1;
    };
    //----------
    // finalization
    h1 ^= len;
    h1 = fmix32(h1);
    return h1;
  }
//-----------------------------------------------------------------------------
//end of hash

__global__
void addinHash(int** Adjacency_partitioned, long* rowcompress, 
    long* dst, int* d_hash, int d_hashSize,
    int numofVertex,int numofEdges){
        
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < numofVertex; 
            i += blockDim.x * gridDim.x) 
         {

            if(i%2 !=0){
                int64_t pair =((int64_t)(i)<<32L);
                uint32_t hval = (hash_murmur(pair)%d_hashSize);
                d_hash[hval]=1;
                // printf(" %d in hash %d, ",i, hval);
            }
         }
}

__global__
void merge(int** Adjacency_partitioned, long* rowcompress, 
    long* dst, int* d_hash, int d_hashSize,
    int numofVertex,int numofEdges, long* d_rowcompress_new, long*d_dst_new){

    // if( threadIdx.x==0) {
        // for (int i = 0; i < numofVertex+1; i++)
        // printf(" %d, ",rowcompress[i]);
        // printf("\n");
        // for (int i = 0; i < numofEdges; i++)
        // printf(" %d, ",dst[i]);
        // printf("\n");
        // for (int i = 0; i < numofEdges; i++)
        // printf(" %d, ",d_dst_new[i]);
        // printf("\n");
        // }

        for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < numofEdges; 
            i += blockDim.x * gridDim.x) 
         {
            int64_t pair =((int64_t)(i)<<32L);
            uint32_t hval = (hash_murmur(pair)%d_hashSize);
            if(d_hash[hval]==0){
                d_dst_new[i]=dst[i];
               
            }
            // else{
            //     printf("%d merged ",i);
            // }
         }
}


int main(int args, char **argv)
{
    std::cout << "Input: ./exe beg csr weight source(optional) report(optional)\n";
    if (args >= 7)
    {
        std::cout << "Wrong input\n";
        return -1;
    }

    int source = 0;
    bool report_nb_vertices_visited = false;

    const char *beg_file = argv[1];
    const char *csr_file = argv[2];
    const char *weight_file = argv[3];

    if (args >= 5)
        source = atoi(argv[4]);
    if (args >= 6)
        report_nb_vertices_visited = atoi(argv[5]);

    auto start = high_resolution_clock::now();
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);

    //template <file_vertex_t, file_index_t, file_weight_t
    //new_vertex_t, new_index_t, new_weight_t>
    graph<long, long, /*int*/ long, long, long, /* char*/ long>
        *ginst = new graph<long, long, /*int*/ long, long, long, /*char*/ long>
        (beg_file, csr_file, weight_file);

    int numofVertex = ginst->vert_count;
    int numofEdges = ginst->edge_count;

    // for (int i = 0; i < numofVertex+1; i++)
    //     cout<<ginst->beg_pos[i]<<",";
    // cout<<"\n";

    // for (int i = 0; i < numofEdges; i++)
    //     cout<<ginst->csr[i]<<",";
    // cout<<"\n";
    
    int **Adjacency = new int *[numofVertex];
    CreateAdjacency(Adjacency, numofVertex, ginst);

    // printAdj(Adjacency, numofVertex);


    int **Adjacency_partitioned = new int *[numofVertex];
    long* rowcompress=ginst->beg_pos;
    long* dst=ginst->csr;



    int* d_hash;
    const int HASH_TABLE_SIZE = 10000000;  int d_hashSize=HASH_TABLE_SIZE;
    printf("start malloc\n");

    hipMalloc(&d_hash, HASH_TABLE_SIZE*sizeof(int));
    hipMemset(d_hash,0,HASH_TABLE_SIZE*sizeof(int));


    int thread=256;
    int block = numofVertex/256 +1;
    cout<<block<<" "<<thread<<endl;
    addinHash<<<block,thread>>> (Adjacency_partitioned,rowcompress,dst,
                                d_hash,d_hashSize, numofVertex,numofEdges);

    
    ////-------------------------test addinHash success
    // int count=0; 
    // int* h_hash=(int*)malloc(HASH_TABLE_SIZE * sizeof(int));
    // hipMemcpy(h_hash,d_hash,HASH_TABLE_SIZE*sizeof(int),hipMemcpyDeviceToHost);
    // for(int i=0; i<d_hashSize; i++){
    //     if (h_hash[i]==1)
    //     count++;
    // }
    // cout<<count;
    ////-------------------------test addinHash success

    long* d_rowcompress; long*  d_dst;
    
    hipMalloc(&d_rowcompress, (numofVertex+1)*sizeof(long));
    hipMalloc(&d_dst, (numofEdges)*sizeof(long));
    hipMemcpy(d_rowcompress,rowcompress,(numofVertex+1)*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(d_dst,dst,numofEdges*sizeof(long),hipMemcpyHostToDevice);

    /// output
    long* d_rowcompress_new; long*  d_dst_new;
    hipMalloc(&d_rowcompress_new, (numofVertex+1)*sizeof(long));
    hipMalloc(&d_dst_new, (numofEdges)*sizeof(long));

    hipMemset(d_rowcompress_new, INF ,(numofVertex+1)*sizeof(long));
    hipMemset(d_dst_new, INF , (numofEdges)*sizeof(long) );



    block = numofEdges/256 +1; //numofEdges
    cout<<block<<" "<<thread<<endl;


    start = high_resolution_clock::now();

    merge<<<block,thread>>> (Adjacency_partitioned,d_rowcompress, d_dst,
        d_hash,d_hashSize, numofVertex,numofEdges,d_rowcompress_new,d_dst_new);
        hipDeviceSynchronize();

    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);
    std::cout << "\n   merge<<<block,thread>>>  " << duration.count() << "\n ";

    auto time1=duration.count();

    //-------------------------test merge success
    int count=0; 
    int* h_dst_new=(int*)malloc(numofEdges*sizeof(long));
    hipMemcpy(h_dst_new,d_dst_new,numofEdges*sizeof(long),hipMemcpyDeviceToHost);
    for(int i=0; i<numofEdges; i++){
        
        if (h_dst_new[i]!=1061109567 && h_dst_new[i]>0  ){
            // printf("%d ",h_dst_new[i]);
            count++;
        }
    }

    for(int i=0; i<(numofEdges<100?numofEdges:100); i++)
        cout<<" "<<h_dst_new[i]<<" ";
        cout<<"\n";

    cout<<"\n"<<numofEdges<<endl;
    cout<<count<<endl;
    cout<<time1;
    //-------------------------test merge success

// hipMalloc(&d_src, HASH_TABLE_SIZE*sizeof(vid_t));
// hipMalloc(&d_dest, HASH_TABLE_SIZE*sizeof(vid_t));
// hipMalloc(&d_hash, HASH_TABLE_SIZE*sizeof(int));














































    return 0;
}




//     int **Adjacency = new int *[numofVertex];
//     CreateAdjacency(Adjacency, numofVertex, ginst);
//     Adjacency[0][0] = 1;
//     Adjacency[1][0] = 1;
//     Adjacency[2][0] = 1;
//     Adjacency[3][0] = 1;
//     // printAdj(Adjacency, numofVertex);
//     initAdj(Adjacency, numofVertex);
//     // printAdj(Adjacency, numofVertex);

//     start = high_resolution_clock::now();
//     ///////////
//     // int *visit = dfs_by_vertexid_array(Adjacency, 0, numofVertex);

//     int *visit = dfs_by_vertexid_array(Adjacency, source, numofVertex, report_nb_vertices_visited);
//     // printlist(visit, numofVertex);
//     ///////////
//     stop = high_resolution_clock::now();
//     duration = duration_cast<milliseconds>(stop - start);
//     std::cout << "\nAverage time SC15 dfs_by_vertexid_array time spends (ms): \n"
//               << duration.count() << endl;
// float time1=duration.count(), time2;

//     initAdj(Adjacency, numofVertex);
//     start = high_resolution_clock::now();
//     ///////////
//     // DFS_optimize5_removeGraphStruct(Adjacency, 0, numofVertex); DFS_optimize5_removeGraphStruct
//     DFS_optimize5_removeGraphStruct(Adjacency, source, numofVertex, report_nb_vertices_visited);
//     ///////////
//     stop = high_resolution_clock::now();
//     duration = duration_cast<milliseconds>(stop - start);
//     std::cout << "\n DFS_optimize5_removeGraphStruct time spends (ms): \n" 
//     << duration.count() << endl;
// time2=duration.count();
//     /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void DFS_visit_removeGraph(int **Adjacency, int start, int &NumDiscover)
{ //   //7/15/2020 removeGraph from DFS_visit_continue_removeSTLstack
    Stack<int> stack(1000);

    if (Adjacency[start][0] == White)
    {
        NumDiscover++;
    }
    if (Adjacency[start][0] == Black)
    {
        return;
    }

    Adjacency[start][0] = Gray;
    stack.push(start);
    while (!stack.isEmpty())
    {
        int currValue = 0;
        stack.pop(&currValue);
        int currColor = Adjacency[start][0];

        ///////////////////////////////////////// ///////////////////////////////////////// TODO pop and push has problem
        //cout << currValue<< "poped, stack.size() " << stack.size() << endl;
        //cout << " at: " << currValue;
        if (Adjacency[currValue][1] == 0)
        {
            continue;
        }
        for (int i = 2; i < Adjacency[currValue][1] + 2; i++)
        {
            //Vertex u = g.vertexs[neighbors[i]];
            int Uvalue = Adjacency[currValue][i];
            int Ucolor = Adjacency[Uvalue][0];
            //int Uvalue = g.vertexs[g.vertexs[currValue].neighbors[i]].value;
            //if (i == neighbors.size() - 1 && u.color == Gray)                ///////////////////////////////////////// never  called on toy graph
            if (i == Adjacency[currValue][1] - 1 && Ucolor == Gray)
            {
                //g.vertexs[currValue].color = Black;
                Adjacency[currValue][0] = Black;
                //cout << " Black: " << currValue;              /////////////////////////////////////////
            }
            if (Ucolor == White)
            {
                //cout << " discover: " << Uvalue;                  /////////////////////////////////////////
                //Ucolor = Gray;
                //set_color(g, u, 1);
                Adjacency[Uvalue][0] = Gray;
                stack.push(Uvalue);
                NumDiscover++;
                continue; // if continue search all neighbors, became  DFS break
            }
        }
        //cout << "\n";                                  /////////////////////////////////////////
    }
}

void DFS_optimize5_removeGraphStruct(int **Adjacency, int start, int numofVertex, bool report_nb_vertices_visited)
{                        //7/15/2020 removeGraphStruct, other same as optimize4
    int NumDiscover = 0; //////////use int* in mutil thread or use
    for (int i = start; i < numofVertex; i++)
    {
        if (Adjacency[i][0] == White)
        {
            DFS_visit_removeGraph(Adjacency, i, NumDiscover);
        }
        if (NumDiscover >= numofVertex)
        {
            if (report_nb_vertices_visited)
                cout << "DFS_optimize5_removeGraphStruct NumDiscover: " << NumDiscover << endl;
            return;
        }
    } //cout << "g.NumDiscover: " << g.NumDiscover << endl;
    if (report_nb_vertices_visited)
        cout << "DFS_optimize5_removeGraphStruct NumDiscover: " << NumDiscover << endl;
}

template <class Value>
Value *mynew_array(size_t nb)
{
    Value *res = (Value *)malloc(size_t(sizeof(Value)) * nb);
    if (res == NULL)
        cout << "mynew_array returned NULL";
    return res;
}
template <class Number, class Size>
void fill_array_seq(Number *array, Size sz, Number val)
{
    memset(array, val, sz * sizeof(Number));
    // for (Size i = Size(0); i < sz; i++)
    //   array[i] = val;
}
static inline void myfree(void *p)
{
    free(p);
}

int *dfs_by_vertexid_array(int **Adjacency, int start, int numofVertex, bool report_nb_vertices_visited)
{
    long nb_vertices_visited = 1;
    typedef int vtxid_type;
    int *visited;
    vtxid_type nb_vertices = numofVertex;
    auto source = start;
    visited = mynew_array<int>(nb_vertices);
    fill_array_seq(visited, nb_vertices, 0); // init all color as 0, means unvisited
    cout << "finish init! ALGO_PHASE:\n";
    vtxid_type *frontier = mynew_array<vtxid_type>(nb_vertices);
    vtxid_type frontier_size = 0;
    frontier[frontier_size++] = source;
    visited[source] = 1;

    while (frontier_size > 0)
    {
        vtxid_type vertex = frontier[--frontier_size];
        vtxid_type degree = Adjacency[vertex][1];
        // vtxid_type *neighbors = graph.adjlists[vertex].get_out_neighbors();
        // cout<<"\nat: "<<vertex<<" degree"<<degree<<" visit: ";

        for (vtxid_type edge = 2; edge < degree + 2; edge++)
        {
            // cout<<Adjacency[vertex][edge]<<" ";
            vtxid_type other = Adjacency[vertex][edge];
            if (visited[other])
                continue;
            if (report_nb_vertices_visited)
                (nb_vertices_visited)++;

            visited[other] = 1;
            frontier[frontier_size++] = other;
        }
    }
    if (report_nb_vertices_visited)
        cout << "dfs_by_vertexid_array nb_vertices_visited： " << nb_vertices_visited << endl;
    myfree(frontier);
    return visited;
}
